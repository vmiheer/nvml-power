#include "nvmlPower.hpp"
#include <cstdlib>
#include <string>
#include <cassert>
#include <unistd.h>
using namespace std;

/*
These may be encompassed in a class if desired. Trivial CUDA programs written for the purpose of benchmarking might prefer this approach.
*/
bool pollThreadStatus = false;
unsigned int deviceCount = 0;
char deviceNameStr[64];

nvmlReturn_t nvmlResult;
nvmlDevice_t nvmlDeviceID;
nvmlPciInfo_t nvmPCIInfo;
nvmlEnableState_t pmmode;
nvmlComputeMode_t computeMode;

pthread_t powerPollThread;
FILE *fp;

FILE *getPowerLogfile() {
  return fp;
}

/*
Poll the GPU using nvml APIs.
*/
void *powerPollingFunc(void *ptr)
{

	unsigned int powerLevel = 0;
	timespec t;
	pthread_setcancelstate(PTHREAD_CANCEL_DISABLE, 0);
	while (pollThreadStatus)
	{

		// Get the power management mode of the GPU.
		nvmlResult = nvmlDeviceGetPowerManagementMode(nvmlDeviceID, &pmmode);

		// The following function may be utilized to handle errors as needed.
		getNVMLError(nvmlResult);

		// Check if power management mode is enabled.
		if (pmmode == NVML_FEATURE_ENABLED)
		{
			// Get the power usage in milliWatts.
			nvmlResult = nvmlDeviceGetPowerUsage(nvmlDeviceID, &powerLevel);
		}

		// The output file stores power in Watts.
		if (clock_gettime(CLOCK_REALTIME, &t) == 0) {
		  // cout << t.tv_sec << t.tv_nsec << endl;
		} else {
		        t = {};
		}
		fprintf(fp, "%lld%.9ld %.3lf\n", t.tv_sec, t.tv_nsec, (powerLevel)/1000.0);
	}
	pthread_setcancelstate(PTHREAD_CANCEL_ENABLE, 0);
	if (clock_gettime(CLOCK_REALTIME, &t) == 0) {
	  // cout << t.tv_sec << t.tv_nsec << endl;
	} else {
	  t = {};
	}
	fprintf(fp, "# end: %lld%.9ld\n", t.tv_sec, t.tv_nsec);

	fclose(fp);
	pthread_exit(0);
}

/*
Start power measurement by spawning a pthread that polls the GPU.
Function needs to be modified as per usage to handle errors as seen fit.
*/
void nvmlAPIRun()
{

	// Initialize nvml.
	nvmlResult = nvmlInit();
	if (NVML_SUCCESS != nvmlResult)
	{
		printf("NVML Init fail: %s\n", nvmlErrorString(nvmlResult));
		exit(0);
	}

	// Count the number of GPUs available.
	nvmlResult = nvmlDeviceGetCount(&deviceCount);
	if (NVML_SUCCESS != nvmlResult)
	{
		printf("Failed to query device count: %s\n", nvmlErrorString(nvmlResult));
		exit(0);
	}

	// for (i = 0; i < deviceCount; i++)
	// {
		// Get the device ID.
	        const char *visibleDevices = getenv("CUDA_VISIBLE_DEVICES");
		assert(visibleDevices && "Couldn't find any device!\n");
		assert(!(strchr(visibleDevices, ',')) && "We don't support multiple devices!");
		const auto i = stoi(visibleDevices);

		nvmlResult = nvmlDeviceGetHandleByIndex(i, &nvmlDeviceID);
		if (NVML_SUCCESS != nvmlResult)
		{
			printf("Failed to get handle for device %d: %s\n", i, nvmlErrorString(nvmlResult));
			exit(0);
		}

		// Get the name of the device.
		nvmlResult = nvmlDeviceGetName(nvmlDeviceID, deviceNameStr, sizeof(deviceNameStr)/sizeof(deviceNameStr[0]));
		if (NVML_SUCCESS != nvmlResult)
		{
			printf("Failed to get name of device %d: %s\n", i, nvmlErrorString(nvmlResult));
			exit(0);
		}

		// Get PCI information of the device.
		nvmlResult = nvmlDeviceGetPciInfo(nvmlDeviceID, &nvmPCIInfo);
		if (NVML_SUCCESS != nvmlResult)
		{
			printf("Failed to get PCI info of device %d: %s\n", i, nvmlErrorString(nvmlResult));
			exit(0);
		}

		// Get the compute mode of the device which indicates CUDA capabilities.
		nvmlResult = nvmlDeviceGetComputeMode(nvmlDeviceID, &computeMode);
		if (NVML_ERROR_NOT_SUPPORTED == nvmlResult)
		{
			printf("This is not a CUDA-capable device.\n");
		}
		else if (NVML_SUCCESS != nvmlResult)
		{
			printf("Failed to get compute mode for device %i: %s\n", i, nvmlErrorString(nvmlResult));
			exit(0);
		}
		// }

	// This statement assumes that the first indexed GPU will be used.
	// If there are multiple GPUs that can be used by the system, this needs to be done with care.
	// Test thoroughly and ensure the correct device ID is being used.
	// nvmlResult = nvmlDeviceGetHandleByIndex(0, &nvmlDeviceID);

	pollThreadStatus = true;

	const char *message = "Test";
	fp = fopen("Power_data.txt", "w+");
	timespec t;
	if (clock_gettime(CLOCK_REALTIME, &t) == 0) {
	  // cout << t.tv_sec << t.tv_nsec << endl;
	} else {
	  t = {};
	}
	fprintf(fp, "# start: %lld%.9ld\n", t.tv_sec, t.tv_nsec);

	int iret = pthread_create(&powerPollThread, NULL, powerPollingFunc, (void*) message);

	if (iret)
	{
		fprintf(stderr,"Error - pthread_create() return code: %d\n",iret);
		exit(0);
	}
}

/*
End power measurement. This ends the polling thread.
*/
void nvmlAPIEnd()
{
	pollThreadStatus = false;
	pthread_join(powerPollThread, NULL);

	nvmlResult = nvmlShutdown();
	if (NVML_SUCCESS != nvmlResult)
	{
		printf("Failed to shut down NVML: %s\n", nvmlErrorString(nvmlResult));
		exit(0);
	}
}

/*
Return a number with a specific meaning. This number needs to be interpreted and handled appropriately.
*/
int getNVMLError(nvmlReturn_t resultToCheck)
{
	if (resultToCheck == NVML_ERROR_UNINITIALIZED)
		return 1;
	if (resultToCheck == NVML_ERROR_INVALID_ARGUMENT)
		return 2;
	if (resultToCheck == NVML_ERROR_NOT_SUPPORTED)
		return 3;
	if (resultToCheck == NVML_ERROR_NO_PERMISSION)
		return 4;
	if (resultToCheck == NVML_ERROR_ALREADY_INITIALIZED)
		return 5;
	if (resultToCheck == NVML_ERROR_NOT_FOUND)
		return 6;
	if (resultToCheck == NVML_ERROR_INSUFFICIENT_SIZE)
		return 7;
	if (resultToCheck == NVML_ERROR_INSUFFICIENT_POWER)
		return 8;
	if (resultToCheck == NVML_ERROR_DRIVER_NOT_LOADED)
		return 9;
	if (resultToCheck == NVML_ERROR_TIMEOUT)
		return 10;
	if (resultToCheck == NVML_ERROR_IRQ_ISSUE)
		return 11;
	if (resultToCheck == NVML_ERROR_LIBRARY_NOT_FOUND)
		return 12;
	if (resultToCheck == NVML_ERROR_FUNCTION_NOT_FOUND)
		return 13;
	if (resultToCheck == NVML_ERROR_CORRUPTED_INFOROM)
		return 14;
	if (resultToCheck == NVML_ERROR_GPU_IS_LOST)
		return 15;
	if (resultToCheck == NVML_ERROR_UNKNOWN)
		return 16;

	return 0;
}
